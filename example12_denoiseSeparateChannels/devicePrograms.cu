#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <hip/hip_runtime.h>

#include "LaunchParams.h"
#include "gdt/random/random.h"

using namespace osc;

#define NUM_LIGHT_SAMPLES 5//��Դ����
#define MAXBOUNCE 5;

namespace osc {

  typedef gdt::LCG<16> Random;
  extern "C" __constant__ LaunchParams optixLaunchParams;
  extern "C" __constant__ float PI = 3.1415926;
  struct PRD {
    Random random;
    vec3f  pixelColor;
    vec3f  pixelNormal;
    vec3f  pixelAlbedo;
	bool done;
	bool first;
	vec3f origin;
	vec3f direction;
	vec3f attenuation;
  };
  static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, vec3f &p)
  {
	  // Uniformly sample disk.
	  const float r = u1;
	  const float phi = 2.0f * PI * u2;
	  p.x = r * gdt::cos(phi);
	  p.y = r * gdt::sin(phi);
	  p.z = sqrtf(fmaxf(0.0f, 1.0f - p.x*p.x - p.y*p.y));
  }//������תΪֱ������
  struct Onb
  {
	  __forceinline__ __device__ Onb(const vec3f &normal)
	  {
		  m_normal = normal;

		  if (fabs(m_normal.x) > fabs(m_normal.z))
		  {
			  m_binormal.x = -m_normal.y;
			  m_binormal.y = m_normal.x;
			  m_binormal.z = 0;
		  }
		  else
		  {
			  m_binormal.x = 0;
			  m_binormal.y = -m_normal.z;
			  m_binormal.z = m_normal.y;
		  }

		  m_binormal = normalize(m_binormal);
		  m_tangent = cross(m_binormal, m_normal);
	  }

	  __forceinline__ __device__ void inverse_transform(vec3f &p) const
	  {
		  p = p.x*m_tangent + p.y*m_binormal + p.z*m_normal;//��p�Ӷ�Ӧ����ϵת������������ϵ
	  }

	  vec3f m_tangent;
	  vec3f m_binormal;
	  vec3f m_normal;//�����໥��ֱ������������һ����������ϵ�Ķ���
  };
  static __forceinline__ __device__
  void *unpackPointer( uint32_t i0, uint32_t i1 )
  {
    const uint64_t uptr = static_cast<uint64_t>( i0 ) << 32 | i1;
    void*           ptr = reinterpret_cast<void*>( uptr ); 
    return ptr;
  }

  static __forceinline__ __device__
  void  packPointer( void* ptr, uint32_t& i0, uint32_t& i1 )
  {
    const uint64_t uptr = reinterpret_cast<uint64_t>( ptr );
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
  }

  template<typename T>
  static __forceinline__ __device__ T *getPRD()
  { 
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>( unpackPointer( u0, u1 ) );
  }
  extern "C" __constant__ float default_F0 = 0.04;
  extern "C" __constant__ float default_alpha = 0.5;
  static __forceinline__ __device__
	 vec3f  half_vector(vec3f view, vec3f light){
	  vec3f half = gdt::normalize(view + light);
	  return half;
  }
  static __forceinline__ __device__
	  vec3f specialmatch(vec3f a, vec3f b) {
	  vec3f output;
	  output.x = a.x * b.x;
	  output.y = a.y * b.y;
	  output.z = a.z * b.z;
	  return output;
  }
  static __forceinline__ __device__
	float NDF(vec3f normal, vec3f half, float alpha){
	  float a2 = alpha * alpha;
	  float NdotH = gdt::dot(normal, half);
	  if (NdotH < 0.0) NdotH = 0.0;
	  float NdotH2 = NdotH * NdotH;
	  float nom = a2;
	  float denom = (NdotH2 * (a2 - 1.0) + 1.0);
	  denom = PI * denom * denom;
	  return nom / denom;
  }
  static __forceinline__ __device__
	float GeometrySchlickGGX(float NdotV, float k){
	  float nom = NdotV;
	  float denom = NdotV * (1.0 - k) + k;
	  return nom / denom;
  }
  static __forceinline__ __device__
	  float K_IBL(float alpha) {
	  float k = alpha * alpha / 2;
	  return k;
  }
  static __forceinline__ __device__
	float GeometrySmith(vec3f normal, vec3f view, vec3f light, float alpha){
	  float k = K_IBL(alpha);
	  float NdotV = gdt::dot(normal, view);
	  if(NdotV < 0.0) NdotV = 0.0;
	  float NdotL = gdt::dot(normal, light);
	  if(NdotL < 0.0) NdotL = 0.0;
	  float ggx1 = GeometrySchlickGGX(NdotV, k);
	  float ggx2 = GeometrySchlickGGX(NdotL, k);
	  return ggx1 * ggx2;
  }
  static __forceinline__ __device__
	vec3f fresnelSchlick(vec3f half, vec3f view, vec3f F0){
	  float HdotV = gdt::dot(half, view);
	  return F0 + (vec3f(1.0) - F0) * (float)pow(1.0 - HdotV, 5.0);
  }
  static __forceinline__ __device__
	 vec3f BRDF(vec3f view, vec3f light, vec3f normal, vec3f F0, float alpha, vec3f color, vec3f kd, vec3f ks) {
	  vec3f v = gdt::normalize(view);
	  vec3f l = gdt::normalize(light);
	  vec3f n = gdt::normalize(normal);
	  vec3f h = half_vector(v, l);
	  vec3f former = specialmatch(color, kd) / PI;
	  float D = NDF(n, h, alpha);
	  float G = GeometrySmith(n, v, l, alpha);
	  vec3f F = fresnelSchlick(h, v, F0);
	  float VdotN = gdt::dot(v, n);
	  float LdotN = gdt::dot(l, n);
	  vec3f latter = specialmatch(D * G * F, ks) / (4 * VdotN * LdotN);
	  return former + latter;
  }


  
  extern "C" __global__ void __closesthit__shadow()
  {
    /* not going to be used ... */
  }
  
  extern "C" __global__ void __closesthit__radiance()
  {
    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
    PRD &prd = *getPRD<PRD>();
	const int   primID = optixGetPrimitiveIndex();
	const vec3i index = sbtData.index[primID];
	const float u = optixGetTriangleBarycentrics().x;
	const float v = optixGetTriangleBarycentrics().y;
	vec3f Kd;
	if (sbtData.Kd == vec3f(0.0)) Kd = vec3f(1.0);
	else Kd = sbtData.Kd;
	const vec3f Ks = sbtData.Ks;
	const vec3f &A = sbtData.vertex[index.x];
	const vec3f &B = sbtData.vertex[index.y];
	const vec3f &C = sbtData.vertex[index.z];
	vec3f Ng = gdt::cross(B - A, C - A);
	vec3f Ns = (sbtData.normal)
		? ((1.f - u - v) * sbtData.normal[index.x]
			+ u * sbtData.normal[index.y]
			+ v * sbtData.normal[index.z])
		: Ng;
	const vec3f rayDir = optixGetWorldRayDirection();

	if (dot(rayDir, Ng) > 0.f) Ng = -Ng;
	Ng = normalize(Ng);

	if (dot(Ng, Ns) < 0.f)
		Ns -= 2.f*dot(Ng, Ns)*Ng;
	Ns = normalize(Ns);
	vec3f diffuseColor = sbtData.color;
	if (sbtData.hasTexture && sbtData.texcoord) {
		const vec2f tc
			= (1.f - u - v) * sbtData.texcoord[index.x]
			+ u * sbtData.texcoord[index.y]
			+ v * sbtData.texcoord[index.z];

		vec4f fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
		diffuseColor *= (vec3f)fromTexture;
	}
	vec3f dirlight = 0;
	const vec3f surfPos
		= (1.f - u - v) * sbtData.vertex[index.x]
		+ u * sbtData.vertex[index.y]
		+ v * sbtData.vertex[index.z];
	const vec3f view = gdt::normalize(optixLaunchParams.camera.position - surfPos);
	const int numLightSamples = NUM_LIGHT_SAMPLES;
	for (int lightSampleID = 0; lightSampleID < numLightSamples; lightSampleID++) {
		const vec3f lightPos
			= optixLaunchParams.light.origin
			+ prd.random() * optixLaunchParams.light.du
			+ prd.random() * optixLaunchParams.light.dv;
		vec3f lightDir = lightPos - surfPos;
		float lightDist = gdt::length(lightDir);
		lightDir = normalize(lightDir);
		const float area = gdt::length(gdt::cross(optixLaunchParams.light.du, optixLaunchParams.light.dv));
		const vec3f Nl = gdt::normalize(gdt::cross(optixLaunchParams.light.du, optixLaunchParams.light.dv));
		const float NldotL = fabs(gdt::dot(Nl, lightDir));
		vec3f fr = BRDF(view, lightDir, Ns, vec3f(default_F0), default_alpha, diffuseColor, Kd, Ks);
		const float NdotL = dot(lightDir, Ns);
		if (NdotL >= 0.f && NldotL >= 0.f) {
			vec3f lightVisibility = 0.f;
			uint32_t u0, u1;
			packPointer(&lightVisibility, u0, u1);
			optixTrace(optixLaunchParams.traversable,
				surfPos + 1e-3f * Ng,
				lightDir,
				1e-3f,      // tmin
				lightDist * (1.f - 1e-3f),  // tmax
				0.0f,       // rayTime
				OptixVisibilityMask(255),
				OPTIX_RAY_FLAG_DISABLE_ANYHIT
				| OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT
				| OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
				SHADOW_RAY_TYPE,            // SBT offset
				RAY_TYPE_COUNT,               // SBT stride
				SHADOW_RAY_TYPE,            // missSBTIndex 
				u0, u1);
			dirlight
				+= lightVisibility
				* optixLaunchParams.light.power
				*  specialmatch(diffuseColor, fr)
				*  (NdotL * NldotL * area / (lightDist*lightDist*numLightSamples));
		}
	}
	prd.pixelColor += dirlight;
	const float distance = gdt::length(optixLaunchParams.light.origin - surfPos);
	vec3f random_sample = -Ns;
	while (gdt::dot(Ns, random_sample) <= 0) {
		float z1 = prd.random();
		float z2 = prd.random();
		cosine_sample_hemisphere(z1, z2, random_sample);
		Onb onb(Ns);
		onb.inverse_transform(random_sample);
	}
	vec3f lightDir = normalize(random_sample);
	prd.direction = lightDir;
	prd.origin = surfPos;
	vec3f fr = BRDF(view, lightDir, Ns, vec3f(default_F0), default_alpha, diffuseColor, Kd, Ks);
	float NdotL = gdt::dot(Ns, lightDir);
	prd.attenuation = specialmatch(NdotL * fr, prd.attenuation);
	if (prd.first) {
		prd.pixelAlbedo = diffuseColor;
		prd.pixelNormal = Ns;
		prd.first = false;
	}
  }
  
  extern "C" __global__ void __anyhit__radiance(){}

  extern "C" __global__ void __anyhit__shadow(){}
  
  extern "C" __global__ void __miss__radiance()
  {
    PRD &prd = *getPRD<PRD>();
	prd.pixelColor = vec3f(1.f);
	prd.done = true;
  }

  extern "C" __global__ void __miss__shadow()
  {
    vec3f &prd = *(vec3f*)getPRD<vec3f>();
    prd = vec3f(1.f);
  }

  //------------------------------------------------------------------------------
  // ray gen program - the actual rendering happens in here
  //------------------------------------------------------------------------------
  extern "C" __global__ void __raygen__renderFrame()
  {
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto &camera = optixLaunchParams.camera;
    
    PRD prd;
    prd.random.init(ix+optixLaunchParams.frame.size.x*iy,
                    optixLaunchParams.frame.frameID);
	prd.attenuation = 1.f;
	prd.done = false;
	prd.pixelColor = 0.f;
	prd.first = true;
	prd.pixelNormal = 0.f;
	prd.pixelAlbedo = 0.f;

    uint32_t u0, u1;
    packPointer( &prd, u0, u1 );

    int numPixelSamples = optixLaunchParams.numPixelSamples;

    vec3f pixelColor = 0.f;
    vec3f pixelNormal = 0.f;
    vec3f pixelAlbedo = 0.f;
	int upperbound = MAXBOUNCE;
    for (int sampleID=0;sampleID<numPixelSamples;sampleID++) {
      vec2f screen(vec2f(ix+prd.random(),iy+prd.random())
                   / vec2f(optixLaunchParams.frame.size));
      vec3f rayDir = normalize(camera.direction
                               + (screen.x - 0.5f) * camera.horizontal
                               + (screen.y - 0.5f) * camera.vertical);
	  vec3f rayOrigin = camera.position;
	  int depth = 0;
	  vec3f result = 0.f;
	  while (!prd.done && depth <= upperbound) {
		  optixTrace(optixLaunchParams.traversable,
			  rayOrigin,
			  rayDir,
			  0.f,    // tmin
			  1e20f,  // tmax
			  0.0f,   // rayTime
			  OptixVisibilityMask(255),
			  OPTIX_RAY_FLAG_NONE,
			  RADIANCE_RAY_TYPE,            // SBT offset
			  RAY_TYPE_COUNT,               // SBT stride
			  RADIANCE_RAY_TYPE,            // missSBTIndex 
			  u0, u1);
		  rayOrigin = prd.origin;
		  rayDir = prd.direction;
		  result += prd.attenuation * prd.pixelColor;
		  depth++;
	  }
      pixelColor  += result;
      pixelNormal += prd.pixelNormal;
      pixelAlbedo += prd.pixelAlbedo;
    }

    vec4f rgba(pixelColor/numPixelSamples,1.f);
    vec4f albedo(pixelAlbedo/numPixelSamples,1.f);
    vec4f normal(pixelNormal/numPixelSamples,1.f);

    const uint32_t fbIndex = ix+iy*optixLaunchParams.frame.size.x;
    if (optixLaunchParams.frame.frameID > 0) {
      rgba
        += float(optixLaunchParams.frame.frameID)
        *  vec4f(optixLaunchParams.frame.colorBuffer[fbIndex]);
      rgba /= (optixLaunchParams.frame.frameID+1.f);
    }
    optixLaunchParams.frame.colorBuffer[fbIndex] = (float4)rgba;
    optixLaunchParams.frame.albedoBuffer[fbIndex] = (float4)albedo;
    optixLaunchParams.frame.normalBuffer[fbIndex] = (float4)normal;
  }
  
} // ::osc
